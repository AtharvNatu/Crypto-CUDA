#include "hip/hip_runtime.h"
// Standard Headers
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <cstring>

// CUDA Headers
#include <hip/hip_runtime.h>
#include "helper_timer.h"

using namespace std;

// Macros
#define BYTE unsigned char
#define LENGTH 16
#define MAX 256
#define FAILURE -1
#define SUCCESS 1

// Variable Declarations
class aes_block
{
    public:
        BYTE block[LENGTH];
};

BYTE sbox[] =
{   /*0    1    2    3    4    5    6    7    8    9    a    b    c    d    e    f */
    0x63,0x7c,0x77,0x7b,0xf2,0x6b,0x6f,0xc5,0x30,0x01,0x67,0x2b,0xfe,0xd7,0xab,0x76, /*0*/ 
    0xca,0x82,0xc9,0x7d,0xfa,0x59,0x47,0xf0,0xad,0xd4,0xa2,0xaf,0x9c,0xa4,0x72,0xc0, /*1*/
    0xb7,0xfd,0x93,0x26,0x36,0x3f,0xf7,0xcc,0x34,0xa5,0xe5,0xf1,0x71,0xd8,0x31,0x15, /*2*/
    0x04,0xc7,0x23,0xc3,0x18,0x96,0x05,0x9a,0x07,0x12,0x80,0xe2,0xeb,0x27,0xb2,0x75, /*3*/
    0x09,0x83,0x2c,0x1a,0x1b,0x6e,0x5a,0xa0,0x52,0x3b,0xd6,0xb3,0x29,0xe3,0x2f,0x84, /*4*/
    0x53,0xd1,0x00,0xed,0x20,0xfc,0xb1,0x5b,0x6a,0xcb,0xbe,0x39,0x4a,0x4c,0x58,0xcf, /*5*/
    0xd0,0xef,0xaa,0xfb,0x43,0x4d,0x33,0x85,0x45,0xf9,0x02,0x7f,0x50,0x3c,0x9f,0xa8, /*6*/ 
    0x51,0xa3,0x40,0x8f,0x92,0x9d,0x38,0xf5,0xbc,0xb6,0xda,0x21,0x10,0xff,0xf3,0xd2, /*7*/
    0xcd,0x0c,0x13,0xec,0x5f,0x97,0x44,0x17,0xc4,0xa7,0x7e,0x3d,0x64,0x5d,0x19,0x73, /*8*/
    0x60,0x81,0x4f,0xdc,0x22,0x2a,0x90,0x88,0x46,0xee,0xb8,0x14,0xde,0x5e,0x0b,0xdb, /*9*/
    0xe0,0x32,0x3a,0x0a,0x49,0x06,0x24,0x5c,0xc2,0xd3,0xac,0x62,0x91,0x95,0xe4,0x79, /*a*/
    0xe7,0xc8,0x37,0x6d,0x8d,0xd5,0x4e,0xa9,0x6c,0x56,0xf4,0xea,0x65,0x7a,0xae,0x08, /*b*/
    0xba,0x78,0x25,0x2e,0x1c,0xa6,0xb4,0xc6,0xe8,0xdd,0x74,0x1f,0x4b,0xbd,0x8b,0x8a, /*c*/
    0x70,0x3e,0xb5,0x66,0x48,0x03,0xf6,0x0e,0x61,0x35,0x57,0xb9,0x86,0xc1,0x1d,0x9e, /*d*/
    0xe1,0xf8,0x98,0x11,0x69,0xd9,0x8e,0x94,0x9b,0x1e,0x87,0xe9,0xce,0x55,0x28,0xdf, /*e*/
    0x8c,0xa1,0x89,0x0d,0xbf,0xe6,0x42,0x68,0x41,0x99,0x2d,0x0f,0xb0,0x54,0xbb,0x16  /*f*/
};

FILE* encFile = NULL;
FILE* decFile = NULL;

ifstream input_file_stream, key_file_stream;

int threads_per_block;
int block_number;
int num_zero_pending;
int input_file_length;

aes_block* aes_block_array = NULL;
aes_block* cuda_aes_block_array = NULL;
BYTE *cuda_key = NULL;

BYTE key[LENGTH * (14 + 1)];
int key_length = 0;
int block_length = LENGTH;
int expanded_key_length;

char temp[LENGTH];

hipDeviceProp_t dev_prop;
hipError_t result = hipSuccess;

// Code
void print_cuda_device_properties(void)
{
	// Code
	printf("\nCUDA INFORMATION : \n");
	printf("\n**************************************************************************************************\n");

	hipError_t ret_cuda_rt;
	int dev_count;

	ret_cuda_rt = hipGetDeviceCount(&dev_count);

	if (ret_cuda_rt != hipSuccess)
		printf("\nCUDA Runtime API Error - hipGetDeviceCount() Failed Due To %s\n", hipGetErrorString(ret_cuda_rt));

	else if (dev_count == 0)
	{
		printf("\nNo CUDA Supported Devices Found On This System ... Exiting !!!\n");
		return;
	}

	else
	{
		printf("Total Number Of CUDA Supporting GPU Device/Devices On This System : %d\n", dev_count);

		for (int i = 0; i < dev_count; i++)
		{
			int driverVersion = 0, runtimeVersion = 0;

			ret_cuda_rt = hipGetDeviceProperties(&dev_prop, i);

			if (ret_cuda_rt != hipSuccess)
			{
				printf("%s in %s at line %d\n", hipGetErrorString(ret_cuda_rt), __FILE__, __LINE__);
				return;
			}

			printf("\n");

			hipDriverGetVersion(&driverVersion);
			hipRuntimeGetVersion(&runtimeVersion);

			printf("================================================================================================\n");
			printf("***** CUDA DRIVER AND RUNTIME INFORMATION *****\n");
			printf("================================================================================================\n");
			printf("CUDA Driver Version					: %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
			printf("CUDA Runtime Version					: %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);
			printf("\n");
			printf("================================================================================================\n");

			printf("***** GPU DEVICE GENERAL INFORMATION *****\n");
			printf("================================================================================================\n");
			printf("GPU Device Number					: %d\n", i);
			printf("GPU Device Name						: %s\n", dev_prop.name);
			printf("GPU Device Compute Compatibility			: %d.%d\n", dev_prop.major, dev_prop.minor);
			printf("GPU Device Clock Rate					: %d\n", dev_prop.clockRate);
			printf("GPU Device Type						: %s", dev_prop.integrated ? "Integrated (On-Board)\n" : "Discrete (Card)\n");
			printf("\n");
			printf("================================================================================================\n");

			printf("***** GPU DEVICE MEMORY INFORMATION *****\n");
			printf("================================================================================================\n");
			printf("GPU Device Total Memory					: %.0f GB = %.0f MB = %llu Bytes\n", ((float)dev_prop.totalGlobalMem / 1048576.0f) / 1024.0f, (float)dev_prop.totalGlobalMem / 1048576.0f, (unsigned long long)dev_prop.totalGlobalMem);
			printf("GPU Device Constant Memory				: %lu Bytes\n", (unsigned long)dev_prop.totalConstMem);
			printf("GPU Device Shared Memory Per SMProcessor		: %lu\n", (unsigned long)dev_prop.sharedMemPerBlock);
			printf("\n");
			printf("================================================================================================\n");

			printf("***** GPU DEVICE MULTIPROCESSOR INFORMATION *****\n");
			printf("================================================================================================\n");
			printf("GPU Device Number Of SMProcessors			: %d\n", dev_prop.multiProcessorCount);
			printf("GPU Device Number Of Registers Per SMProcessor		: %d\n", dev_prop.regsPerBlock);
			printf("\n");
			printf("================================================================================================\n");

			printf("***** GPU DEVICE THREAD INFORMATION *****\n");
			printf("================================================================================================\n");
			printf("GPU Device Maximum Number Of Threads Per SMProcessor	: %d\n", dev_prop.maxThreadsPerMultiProcessor);
			printf("GPU Device Maximum Number Of Threads Per Block		: %d\n", dev_prop.maxThreadsPerBlock);
			printf("GPU Device Threads In Warp				: %d\n", dev_prop.warpSize);
			printf("GPU Device Maximum Thread Dimensions			: %d, %d, %d\n", dev_prop.maxThreadsDim[0], dev_prop.maxThreadsDim[1], dev_prop.maxThreadsDim[2]);
			printf("GPU Device Maximum Grid Dimensions			: %d, %d, %d\n", dev_prop.maxGridSize[0], dev_prop.maxGridSize[1], dev_prop.maxGridSize[2]);
			printf("\n");
			printf("================================================================================================\n");

			printf("***** GPU DEVICE DRIVER INFORMATION *****\n");
			printf("================================================================================================\n");
			printf("GPU Device Has ECC Support				: %s\n", dev_prop.ECCEnabled ? "Enabled" : "Disabled");

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
			printf("GPU Device CUDA Driver Mode ( TCC Or WDDM )		: %s\n", dev_prop.tccDriver ? "TCC ( Tesla Compute Cluster Driver )" : "WDDM ( Windows Display Driver Model )");
#endif

			printf("================================================================================================\n");

			printf("\n**************************************************************************************************\n");

		}
	}
}

__device__ void sub_bytes(BYTE state[], BYTE sbox[])
{
    for (int i = 0; i < LENGTH; i++)
        state[i] = sbox[state[i]];
}

__device__ void add_round_key(BYTE state[], BYTE round_key[])
{
    for (int i = 0; i < LENGTH; i++)
        state[i] = state[i] ^ round_key[i];
}

__device__ void shift_rows(BYTE state[], BYTE shift_tab[])
{
    BYTE h[LENGTH];
    memcpy(h, state, LENGTH);

    for (int i = 0; i < LENGTH; i++)
        state[i] = h[shift_tab[i]];
}

__device__ void mix_columns(BYTE state[], BYTE time[])
{
    for (int i = 0; i < LENGTH; i = i + 4)
    {
        BYTE s0 = state[i + 0];
        BYTE s1 = state[i + 1];
        BYTE s2 = state[i + 2];
        BYTE s3 = state[i + 3];

        BYTE h = s0 ^ s1 ^ s2 ^ s3;

        state[i + 0] = state[i + 0] ^ h ^ time[s0 ^ s1];
        state[i + 1] = state[i + 1] ^ h ^ time[s1 ^ s2];
        state[i + 2] = state[i + 2] ^ h ^ time[s2 ^ s3];
        state[i + 3] = state[i + 3] ^ h ^ time[s3 ^ s0];
    }
}

__device__ void mix_columns_inverse(BYTE state[], BYTE time[])
{
    for (int i = 0; i < LENGTH; i = i + 4)
    {
        BYTE s0 = state[i + 0];
        BYTE s1 = state[i + 1];
        BYTE s2 = state[i + 2];
        BYTE s3 = state[i + 3];

        BYTE h = s0 ^ s1 ^ s2 ^ s3;
        BYTE xh = time[h];
        BYTE h1 = time[time[xh ^ s0 ^ s2]] ^ h;
        BYTE h2 = time[time[xh ^ s1 ^ s3]] ^ h;

        state[i + 0] = state[i + 0] ^ h1 ^ time[s0 ^ s1];
        state[i + 1] = state[i + 1] ^ h2 ^ time[s1 ^ s2];
        state[i + 2] = state[i + 2] ^ h1 ^ time[s2 ^ s3];
        state[i + 3] = state[i + 3] ^ h2 ^ time[s3 ^ s0];
    }
}

__device__ void aes_init(BYTE sbox[], BYTE shift_row_tab[], BYTE sbox_inverse[], BYTE time[], BYTE shift_row_tab_inverse[])
{
    shift_row_tab[0]=0;
    shift_row_tab[1]=5;
    shift_row_tab[2]=10;
    shift_row_tab[3]=15;
    shift_row_tab[4]=4;
    shift_row_tab[5]=9;
    shift_row_tab[6]=14;
    shift_row_tab[7]=3;
    shift_row_tab[8]=8;
    shift_row_tab[9]=13;
    shift_row_tab[10]=2;
    shift_row_tab[11]=7;
    shift_row_tab[12]=12;
    shift_row_tab[13]=1;
    shift_row_tab[14]=6;
    shift_row_tab[15]=11;

    sbox[0] = 0x63;sbox[1] = 0x7c;sbox[2] = 0x77;sbox[3] = 0x7b;sbox[4] = 0xf2;sbox[5] = 0x6b;sbox[6] = 0x6f;sbox[7] = 0xc5;sbox[8] = 0x30;sbox[9] = 0x1;sbox[10] = 0x67;sbox[11] = 0x2b;sbox[12] = 0xfe;sbox[13] = 0xd7;sbox[14] = 0xab;sbox[15] = 0x76;
    sbox[16] = 0xca;sbox[17] = 0x82;sbox[18] = 0xc9;sbox[19] = 0x7d;sbox[20] = 0xfa;sbox[21] = 0x59;sbox[22] = 0x47;sbox[23] = 0xf0;sbox[24] = 0xad;sbox[25] = 0xd4;sbox[26] = 0xa2;sbox[27] = 0xaf;sbox[28] = 0x9c;sbox[29] = 0xa4;sbox[30] = 0x72;sbox[31] = 0xc0;
    sbox[32] = 0xb7;sbox[33] = 0xfd;sbox[34] = 0x93;sbox[35] = 0x26;sbox[36] = 0x36;sbox[37] = 0x3f;sbox[38] = 0xf7;sbox[39] = 0xcc;sbox[40] = 0x34;sbox[41] = 0xa5;sbox[42] = 0xe5;sbox[43] = 0xf1;sbox[44] = 0x71;sbox[45] = 0xd8;sbox[46] = 0x31;sbox[47] = 0x15;
    sbox[48] = 0x4;sbox[49] = 0xc7;sbox[50] = 0x23;sbox[51] = 0xc3;sbox[52] = 0x18;sbox[53] = 0x96;sbox[54] = 0x5;sbox[55] = 0x9a;sbox[56] = 0x7;sbox[57] = 0x12;sbox[58] = 0x80;sbox[59] = 0xe2;sbox[60] = 0xeb;sbox[61] = 0x27;sbox[62] = 0xb2;sbox[63] = 0x75;
    sbox[64] = 0x9;sbox[65] = 0x83;sbox[66] = 0x2c;sbox[67] = 0x1a;sbox[68] = 0x1b;sbox[69] = 0x6e;sbox[70] = 0x5a;sbox[71] = 0xa0;sbox[72] = 0x52;sbox[73] = 0x3b;sbox[74] = 0xd6;sbox[75] = 0xb3;sbox[76] = 0x29;sbox[77] = 0xe3;sbox[78] = 0x2f;sbox[79] = 0x84;
    sbox[80] = 0x53;sbox[81] = 0xd1;sbox[82] = 0x0;sbox[83] = 0xed;sbox[84] = 0x20;sbox[85] = 0xfc;sbox[86] = 0xb1;sbox[87] = 0x5b;sbox[88] = 0x6a;sbox[89] = 0xcb;sbox[90] = 0xbe;sbox[91] = 0x39;sbox[92] = 0x4a;sbox[93] = 0x4c;sbox[94] = 0x58;sbox[95] = 0xcf;
    sbox[96] = 0xd0;sbox[97] = 0xef;sbox[98] = 0xaa;sbox[99] = 0xfb;sbox[100] = 0x43;sbox[101] = 0x4d;sbox[102] = 0x33;sbox[103] = 0x85;sbox[104] = 0x45;sbox[105] = 0xf9;sbox[106] = 0x2;sbox[107] = 0x7f;sbox[108] = 0x50;sbox[109] = 0x3c;sbox[110] = 0x9f;sbox[111] = 0xa8;
    sbox[112] = 0x51;sbox[113] = 0xa3;sbox[114] = 0x40;sbox[115] = 0x8f;sbox[116] = 0x92;sbox[117] = 0x9d;sbox[118] = 0x38;sbox[119] = 0xf5;sbox[120] = 0xbc;sbox[121] = 0xb6;sbox[122] = 0xda;sbox[123] = 0x21;sbox[124] = 0x10;sbox[125] = 0xff;sbox[126] = 0xf3;sbox[127] = 0xd2;
    sbox[128] = 0xcd;sbox[129] = 0xc;sbox[130] = 0x13;sbox[131] = 0xec;sbox[132] = 0x5f;sbox[133] = 0x97;sbox[134] = 0x44;sbox[135] = 0x17;sbox[136] = 0xc4;sbox[137] = 0xa7;sbox[138] = 0x7e;sbox[139] = 0x3d;sbox[140] = 0x64;sbox[141] = 0x5d;sbox[142] = 0x19;sbox[143] = 0x73;
    sbox[144] = 0x60;sbox[145] = 0x81;sbox[146] = 0x4f;sbox[147] = 0xdc;sbox[148] = 0x22;sbox[149] = 0x2a;sbox[150] = 0x90;sbox[151] = 0x88;sbox[152] = 0x46;sbox[153] = 0xee;sbox[154] = 0xb8;sbox[155] = 0x14;sbox[156] = 0xde;sbox[157] = 0x5e;sbox[158] = 0xb;sbox[159] = 0xdb;
    sbox[160] = 0xe0;sbox[161] = 0x32;sbox[162] = 0x3a;sbox[163] = 0xa;sbox[164] = 0x49;sbox[165] = 0x6;sbox[166] = 0x24;sbox[167] = 0x5c;sbox[168] = 0xc2;sbox[169] = 0xd3;sbox[170] = 0xac;sbox[171] = 0x62;sbox[172] = 0x91;sbox[173] = 0x95;sbox[174] = 0xe4;sbox[175] = 0x79;
    sbox[176] = 0xe7;sbox[177] = 0xc8;sbox[178] = 0x37;sbox[179] = 0x6d;sbox[180] = 0x8d;sbox[181] = 0xd5;sbox[182] = 0x4e;sbox[183] = 0xa9;sbox[184] = 0x6c;sbox[185] = 0x56;sbox[186] = 0xf4;sbox[187] = 0xea;sbox[188] = 0x65;sbox[189] = 0x7a;sbox[190] = 0xae;sbox[191] = 0x8;
    sbox[192] = 0xba;sbox[193] = 0x78;sbox[194] = 0x25;sbox[195] = 0x2e;sbox[196] = 0x1c;sbox[197] = 0xa6;sbox[198] = 0xb4;sbox[199] = 0xc6;sbox[200] = 0xe8;sbox[201] = 0xdd;sbox[202] = 0x74;sbox[203] = 0x1f;sbox[204] = 0x4b;sbox[205] = 0xbd;sbox[206] = 0x8b;sbox[207] = 0x8a;
    sbox[208] = 0x70;sbox[209] = 0x3e;sbox[210] = 0xb5;sbox[211] = 0x66;sbox[212] = 0x48;sbox[213] = 0x3;sbox[214] = 0xf6;sbox[215] = 0xe;sbox[216] = 0x61;sbox[217] = 0x35;sbox[218] = 0x57;sbox[219] = 0xb9;sbox[220] = 0x86;sbox[221] = 0xc1;sbox[222] = 0x1d;sbox[223] = 0x9e;
    sbox[224] = 0xe1;sbox[225] = 0xf8;sbox[226] = 0x98;sbox[227] = 0x11;sbox[228] = 0x69;sbox[229] = 0xd9;sbox[230] = 0x8e;sbox[231] = 0x94;sbox[232] = 0x9b;sbox[233] = 0x1e;sbox[234] = 0x87;sbox[235] = 0xe9;sbox[236] = 0xce;sbox[237] = 0x55;sbox[238] = 0x28;sbox[239] = 0xdf;
    sbox[240] = 0x8c;sbox[241] = 0xa1;sbox[242] = 0x89;sbox[243] = 0xd;sbox[244] = 0xbf;sbox[245] = 0xe6;sbox[246] = 0x42;sbox[247] = 0x68;sbox[248] = 0x41;sbox[249] = 0x99;sbox[250] = 0x2d;sbox[251] = 0xf;sbox[252] = 0xb0;sbox[253] = 0x54;sbox[254] = 0xbb; sbox[255] = 0x16;

    for (int i = 0; i < MAX; i++)
        sbox_inverse[sbox[i]] = i;
    
    for (int i = 0; i < LENGTH; i++)
        shift_row_tab_inverse[shift_row_tab[i]] = i;
    
    for (int i = 0; i < 128; i++)
    {
        time[i] = i << 1;
        time[128 + i] = (i << 1) ^ 0x1b;
    }
}

__device__ void aes_init_inverse(BYTE sbox[], BYTE shift_row_tab[], BYTE sbox_inverse[], BYTE time[], BYTE shift_row_tab_inverse[])
{
    shift_row_tab[0]=0;
    shift_row_tab[1]=5;
    shift_row_tab[2]=10;
    shift_row_tab[3]=15;
    shift_row_tab[4]=4;
    shift_row_tab[5]=9;
    shift_row_tab[6]=14;
    shift_row_tab[7]=3;
    shift_row_tab[8]=8;
    shift_row_tab[9]=13;
    shift_row_tab[10]=2;
    shift_row_tab[11]=7;
    shift_row_tab[12]=12;
    shift_row_tab[13]=1;
    shift_row_tab[14]=6;
    shift_row_tab[15]=11;

    sbox_inverse[0] = 0x52;sbox_inverse[1] = 0x9;sbox_inverse[2] = 0x6a;sbox_inverse[3] = 0xd5;sbox_inverse[4] = 0x30;sbox_inverse[5] = 0x36;sbox_inverse[6] = 0xa5;sbox_inverse[7] = 0x38;sbox_inverse[8] = 0xbf;sbox_inverse[9] = 0x40;sbox_inverse[10] = 0xa3;sbox_inverse[11] = 0x9e;sbox_inverse[12] = 0x81;sbox_inverse[13] = 0xf3;sbox_inverse[14] = 0xd7;sbox_inverse[15] = 0xfb;
    sbox_inverse[16] = 0x7c;sbox_inverse[17] = 0xe3;sbox_inverse[18] = 0x39;sbox_inverse[19] = 0x82;sbox_inverse[20] = 0x9b;sbox_inverse[21] = 0x2f;sbox_inverse[22] = 0xff;sbox_inverse[23] = 0x87;sbox_inverse[24] = 0x34;sbox_inverse[25] = 0x8e;sbox_inverse[26] = 0x43;sbox_inverse[27] = 0x44;sbox_inverse[28] = 0xc4;sbox_inverse[29] = 0xde;sbox_inverse[30] = 0xe9;sbox_inverse[31] = 0xcb;
    sbox_inverse[32] = 0x54;sbox_inverse[33] = 0x7b;sbox_inverse[34] = 0x94;sbox_inverse[35] = 0x32;sbox_inverse[36] = 0xa6;sbox_inverse[37] = 0xc2;sbox_inverse[38] = 0x23;sbox_inverse[39] = 0x3d;sbox_inverse[40] = 0xee;sbox_inverse[41] = 0x4c;sbox_inverse[42] = 0x95;sbox_inverse[43] = 0xb;sbox_inverse[44] = 0x42;sbox_inverse[45] = 0xfa;sbox_inverse[46] = 0xc3;sbox_inverse[47] = 0x4e;
    sbox_inverse[48] = 0x8;sbox_inverse[49] = 0x2e;sbox_inverse[50] = 0xa1;sbox_inverse[51] = 0x66;sbox_inverse[52] = 0x28;sbox_inverse[53] = 0xd9;sbox_inverse[54] = 0x24;sbox_inverse[55] = 0xb2;sbox_inverse[56] = 0x76;sbox_inverse[57] = 0x5b;sbox_inverse[58] = 0xa2;sbox_inverse[59] = 0x49;sbox_inverse[60] = 0x6d;sbox_inverse[61] = 0x8b;sbox_inverse[62] = 0xd1;sbox_inverse[63] = 0x25;
    sbox_inverse[64] = 0x72;sbox_inverse[65] = 0xf8;sbox_inverse[66] = 0xf6;sbox_inverse[67] = 0x64;sbox_inverse[68] = 0x86;sbox_inverse[69] = 0x68;sbox_inverse[70] = 0x98;sbox_inverse[71] = 0x16;sbox_inverse[72] = 0xd4;sbox_inverse[73] = 0xa4;sbox_inverse[74] = 0x5c;sbox_inverse[75] = 0xcc;sbox_inverse[76] = 0x5d;sbox_inverse[77] = 0x65;sbox_inverse[78] = 0xb6;sbox_inverse[79] = 0x92;
    sbox_inverse[80] = 0x6c;sbox_inverse[81] = 0x70;sbox_inverse[82] = 0x48;sbox_inverse[83] = 0x50;sbox_inverse[84] = 0xfd;sbox_inverse[85] = 0xed;sbox_inverse[86] = 0xb9;sbox_inverse[87] = 0xda;sbox_inverse[88] = 0x5e;sbox_inverse[89] = 0x15;sbox_inverse[90] = 0x46;sbox_inverse[91] = 0x57;sbox_inverse[92] = 0xa7;sbox_inverse[93] = 0x8d;sbox_inverse[94] = 0x9d;sbox_inverse[95] = 0x84;
    sbox_inverse[96] = 0x90;sbox_inverse[97] = 0xd8;sbox_inverse[98] = 0xab;sbox_inverse[99] = 0x0;sbox_inverse[100] = 0x8c;sbox_inverse[101] = 0xbc;sbox_inverse[102] = 0xd3;sbox_inverse[103] = 0xa;sbox_inverse[104] = 0xf7;sbox_inverse[105] = 0xe4;sbox_inverse[106] = 0x58;sbox_inverse[107] = 0x5;sbox_inverse[108] = 0xb8;sbox_inverse[109] = 0xb3;sbox_inverse[110] = 0x45;sbox_inverse[111] = 0x6;
    sbox_inverse[112] = 0xd0;sbox_inverse[113] = 0x2c;sbox_inverse[114] = 0x1e;sbox_inverse[115] = 0x8f;sbox_inverse[116] = 0xca;sbox_inverse[117] = 0x3f;sbox_inverse[118] = 0xf;sbox_inverse[119] = 0x2;sbox_inverse[120] = 0xc1;sbox_inverse[121] = 0xaf;sbox_inverse[122] = 0xbd;sbox_inverse[123] = 0x3;sbox_inverse[124] = 0x1;sbox_inverse[125] = 0x13;sbox_inverse[126] = 0x8a;sbox_inverse[127] = 0x6b;
    sbox_inverse[128] = 0x3a;sbox_inverse[129] = 0x91;sbox_inverse[130] = 0x11;sbox_inverse[131] = 0x41;sbox_inverse[132] = 0x4f;sbox_inverse[133] = 0x67;sbox_inverse[134] = 0xdc;sbox_inverse[135] = 0xea;sbox_inverse[136] = 0x97;sbox_inverse[137] = 0xf2;sbox_inverse[138] = 0xcf;sbox_inverse[139] = 0xce;sbox_inverse[140] = 0xf0;sbox_inverse[141] = 0xb4;sbox_inverse[142] = 0xe6;sbox_inverse[143] = 0x73;
    sbox_inverse[144] = 0x96;sbox_inverse[145] = 0xac;sbox_inverse[146] = 0x74;sbox_inverse[147] = 0x22;sbox_inverse[148] = 0xe7;sbox_inverse[149] = 0xad;sbox_inverse[150] = 0x35;sbox_inverse[151] = 0x85;sbox_inverse[152] = 0xe2;sbox_inverse[153] = 0xf9;sbox_inverse[154] = 0x37;sbox_inverse[155] = 0xe8;sbox_inverse[156] = 0x1c;sbox_inverse[157] = 0x75;sbox_inverse[158] = 0xdf;sbox_inverse[159] = 0x6e;
    sbox_inverse[160] = 0x47;sbox_inverse[161] = 0xf1;sbox_inverse[162] = 0x1a;sbox_inverse[163] = 0x71;sbox_inverse[164] = 0x1d;sbox_inverse[165] = 0x29;sbox_inverse[166] = 0xc5;sbox_inverse[167] = 0x89;sbox_inverse[168] = 0x6f;sbox_inverse[169] = 0xb7;sbox_inverse[170] = 0x62;sbox_inverse[171] = 0xe;sbox_inverse[172] = 0xaa;sbox_inverse[173] = 0x18;sbox_inverse[174] = 0xbe;sbox_inverse[175] = 0x1b;
    sbox_inverse[176] = 0xfc;sbox_inverse[177] = 0x56;sbox_inverse[178] = 0x3e;sbox_inverse[179] = 0x4b;sbox_inverse[180] = 0xc6;sbox_inverse[181] = 0xd2;sbox_inverse[182] = 0x79;sbox_inverse[183] = 0x20;sbox_inverse[184] = 0x9a;sbox_inverse[185] = 0xdb;sbox_inverse[186] = 0xc0;sbox_inverse[187] = 0xfe;sbox_inverse[188] = 0x78;sbox_inverse[189] = 0xcd;sbox_inverse[190] = 0x5a;sbox_inverse[191] = 0xf4;
    sbox_inverse[192] = 0x1f;sbox_inverse[193] = 0xdd;sbox_inverse[194] = 0xa8;sbox_inverse[195] = 0x33;sbox_inverse[196] = 0x88;sbox_inverse[197] = 0x7;sbox_inverse[198] = 0xc7;sbox_inverse[199] = 0x31;sbox_inverse[200] = 0xb1;sbox_inverse[201] = 0x12;sbox_inverse[202] = 0x10;sbox_inverse[203] = 0x59;sbox_inverse[204] = 0x27;sbox_inverse[205] = 0x80;sbox_inverse[206] = 0xec;sbox_inverse[207] = 0x5f;
    sbox_inverse[208] = 0x60;sbox_inverse[209] = 0x51;sbox_inverse[210] = 0x7f;sbox_inverse[211] = 0xa9;sbox_inverse[212] = 0x19;sbox_inverse[213] = 0xb5;sbox_inverse[214] = 0x4a;sbox_inverse[215] = 0xd;sbox_inverse[216] = 0x2d;sbox_inverse[217] = 0xe5;sbox_inverse[218] = 0x7a;sbox_inverse[219] = 0x9f;sbox_inverse[220] = 0x93;sbox_inverse[221] = 0xc9;sbox_inverse[222] = 0x9c;sbox_inverse[223] = 0xef;
    sbox_inverse[224] = 0xa0;sbox_inverse[225] = 0xe0;sbox_inverse[226] = 0x3b;sbox_inverse[227] = 0x4d;sbox_inverse[228] = 0xae;sbox_inverse[229] = 0x2a;sbox_inverse[230] = 0xf5;sbox_inverse[231] = 0xb0;sbox_inverse[232] = 0xc8;sbox_inverse[233] = 0xeb;sbox_inverse[234] = 0xbb;sbox_inverse[235] = 0x3c;sbox_inverse[236] = 0x83;sbox_inverse[237] = 0x53;sbox_inverse[238] = 0x99;sbox_inverse[239] = 0x61;
    sbox_inverse[240] = 0x17;sbox_inverse[241] = 0x2b;sbox_inverse[242] = 0x4;sbox_inverse[243] = 0x7e;sbox_inverse[244] = 0xba;sbox_inverse[245] = 0x77;sbox_inverse[246] = 0xd6;sbox_inverse[247] = 0x26;sbox_inverse[248] = 0xe1;sbox_inverse[249] = 0x69;sbox_inverse[250] = 0x14;sbox_inverse[251] = 0x63;sbox_inverse[252] = 0x55;sbox_inverse[253] = 0x21;sbox_inverse[254] = 0xc;sbox_inverse[255] = 0x7d;

    for (int i = 0; i < LENGTH; i++)
        shift_row_tab_inverse[shift_row_tab[i]] = i;
    
    for (int i = 0; i < 128; i++)
    {
        time[i] = i << 1;
        time[128 + i] = (i << 1) ^ 0x1b;
    }
}

__global__ void aes_encrypt(aes_block aes_block_array[], BYTE key[], int key_length, int block_number)
{
    int global_thread_index = blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ BYTE shift_row_tab[LENGTH];
    __shared__ BYTE shift_row_tab_inverse[LENGTH];
    __shared__ BYTE sbox[MAX];
    __shared__ BYTE sbox_inverse[MAX];
    __shared__ BYTE time[MAX];

    if (global_thread_index < block_number)
    {
        if (threadIdx.x == 0)
            aes_init(sbox, shift_row_tab, sbox_inverse, time, shift_row_tab_inverse);
        
        __syncthreads();

        BYTE block[LENGTH];
        for (int i = 0; i < LENGTH; i++)
            block[i] = aes_block_array[global_thread_index].block[i];

        int length = key_length;
        int i;

        add_round_key(block, &key[0]);
        for (i = LENGTH; i < length - LENGTH; i = i + LENGTH)
        {
            sub_bytes(block, sbox);
            shift_rows(block, shift_row_tab);
            mix_columns(block, time);
            add_round_key(block, &key[i]);
        }
        sub_bytes(block, sbox);
        shift_rows(block, shift_row_tab);
        add_round_key(block, &key[i]);

        for (int j = 0; j < LENGTH; j++)
            aes_block_array[global_thread_index].block[i] = block[i];
    }
}

__global__ void aes_decrypt(aes_block aes_block_array[], BYTE key[], int key_length, int block_number)
{
    int global_thread_index = blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ BYTE shift_row_tab[LENGTH];
    __shared__ BYTE shift_row_tab_inverse[LENGTH];
    __shared__ BYTE sbox[MAX];
    __shared__ BYTE sbox_inverse[MAX];
    __shared__ BYTE time[MAX];

    if (global_thread_index < block_number)
    {
        if (threadIdx.x == 0)
            aes_init_inverse(sbox, shift_row_tab, sbox_inverse, time, shift_row_tab_inverse);
        
        __syncthreads();

        BYTE block[LENGTH];
        for (int i = 0; i < LENGTH; i++)
            block[i] = aes_block_array[global_thread_index].block[i];

        int length = key_length;	
        int i;

        add_round_key(block, &key[length - LENGTH]);
        shift_rows(block, shift_row_tab_inverse);
        sub_bytes(block, sbox_inverse);
        for (i = length - 32; i >= LENGTH; i = i - LENGTH)
        {
            add_round_key(block, &key[i]);
            mix_columns_inverse(block, time);
            shift_rows(block, shift_row_tab_inverse);
            sub_bytes(block, sbox_inverse);
        }
        add_round_key(block, &key[0]);

        for (int j = 0; j < LENGTH; j++)
            aes_block_array[global_thread_index].block[i] = block[i];
    }
}

int expand_key(BYTE key[], int key_length)
{
	int length = key_length;
	int Rcon = 1;
	int ks;

	BYTE temp_array_1[4], temp_array_2[4];

	switch(length)
	{
		case 16:
			ks = 16 * (10 + 1);
		break;

		case 24:
			ks = 16 * (12 + 1);
		break;

		case 32:
			ks = 16 * (14 + 1);
		break;

		default:
			printf("Expand Key : Key Lengths Permitted Are -> 16, 24 or 32 Bytes Only !!!\n");
		break;
	}

	for (int i = length; i < ks; i = i + 4)
	{
		memcpy(temp_array_1, &key[i - 4], 4);

		if (i % length == 0)
		{
			temp_array_2[0] = sbox[temp_array_1[1]] ^ Rcon;
			temp_array_2[1] = sbox[temp_array_1[2]];
			temp_array_2[2] = sbox[temp_array_1[3]];
			temp_array_2[3] = sbox[temp_array_1[0]];

			memcpy(temp_array_1, temp_array_2, 4);

			if ((Rcon <<= 1) >= 256)
				Rcon = Rcon ^ 0x11b;
		}

		else if ((length > 24) && (i % length == 16))
		{
			temp_array_2[0] = sbox[temp_array_1[1]];
			temp_array_2[1] = sbox[temp_array_1[2]];
			temp_array_2[2] = sbox[temp_array_1[3]];
			temp_array_2[3] = sbox[temp_array_1[0]];

			memcpy(temp_array_1, temp_array_2, 4);
		}

		for (int j = 0; j < 4; j++)
			key[i + j] = key[i + j - length] ^ temp_array_1[j];
		
	}

	return ks;
}

void print_file_data(BYTE array[], int length, FILE* fp, int file)
{
	// Local Variables
	int flag = 0;

	// Code
	switch(file)
	{
		case 1:
			for (int i = 0; i < length; i++)
				fprintf(fp, "%02x", array[i]);
			fprintf(fp, "\n");
		break;

		case 2:
			for (int i = 0; i < length; i++)
			{
				fprintf(fp, "%c", array[i]);
				if (array[i] == '\n')
					flag++;
			}
		break;

		case 3:
			for (int i = 0; i < length; i++)
			{
				if (array[i] == '\0')
					return;
				fprintf(fp, "%c", array[i]);
				if (array[i] == '\n')
					flag++;
			}
		break;
	}
}

int get_file_data(char* argv[])
{
    // Function Declaration
    void cleanup(void);

    // Code
	input_file_stream.open(argv[1], ifstream::binary);
	if (!input_file_stream)
		return FAILURE;
	input_file_stream.seekg(0, ios::end);
	input_file_length = input_file_stream.tellg();
	input_file_stream.seekg(0, ios::beg);

	block_number = input_file_length / LENGTH;
	num_zero_pending = input_file_length % LENGTH;

	key_file_stream.open(argv[2]);
	while (key_file_stream.peek() != EOF)
	{
		key_file_stream >> key[key_length];
		if (key_file_stream.eof())
			break;
		key_length++;
	}

	switch(key_length)
	{
		case 16:
		case 24:
		case 32:
		break;

		default:
			printf("Key Length Should Be 128, 192 or 256 bits !!!\n");
			return FAILURE;
	}

	expanded_key_length = expand_key(key, key_length);

	if (num_zero_pending != 0)
		aes_block_array = new aes_block[block_number + 1];
	else
		aes_block_array = new aes_block[block_number];
	
	encFile = fopen(argv[3], "wb");
	decFile = fopen(argv[4], "wb");

	for (int i = 0; i < block_number; i++)
	{
		input_file_stream.read(temp, LENGTH);
		for (int j = 0; j < LENGTH; j++)
			aes_block_array[i].block[j] = (unsigned char)temp[j];
	}

	if (num_zero_pending != 0)
	{
		input_file_stream.read(temp, num_zero_pending);
		for (int j = 0; j < LENGTH; j++)
			aes_block_array[block_number].block[j] = (unsigned char)temp[j];
		for (int j = 1; j < LENGTH - num_zero_pending; j++)
			aes_block_array[block_number].block[LENGTH - j] = '\0';
	
		block_number++;
	}

	char* num_threads = argv[5];
	int number_of_threads = atoi(num_threads);

    dim3 ThreadsPerBlock(number_of_threads);
    dim3 BlocksPerGrid(256);

    result = hipMalloc((void **)&cuda_aes_block_array, block_number * sizeof(aes_block));
    if (result != hipSuccess)
	{
		printf("\nDevice Memory Allocation Failed For cuda_aes_block_array ... Exiting Now !!!\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

    result = hipMalloc((void **)&cuda_key, LENGTH * 15 * sizeof(BYTE));
    if (result != hipSuccess)
	{
		printf("\nDevice Memory Allocation Failed For cuda_key ... Exiting Now !!!\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

    result = hipMemcpy(cuda_aes_block_array, aes_block_array, block_number * sizeof(aes_block), hipMemcpyHostToDevice);
    if (result != hipSuccess)
	{
		printf("\nHost To Device Data Copy Failed For cuda_aes_block_array ... Exiting Now !!!\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

    result = hipMemcpy(cuda_key, key, LENGTH * 15 * sizeof(BYTE), hipMemcpyHostToDevice);
    if (result != hipSuccess)
	{
		printf("\nHost To Device Data Copy Failed For cuda_key ... Exiting Now !!!\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

    StopWatchInterface* timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

    aes_encrypt <<< BlocksPerGrid, ThreadsPerBlock >>>(cuda_aes_block_array, cuda_key, expanded_key_length, block_number);

    sdkStopTimer(&timer);
	float timeToEncrypt = sdkGetTimerValue(&timer);
	printf("\nTime To Encrypt Image on %s = %.6f seconds\n", dev_prop.name, timeToEncrypt);
	sdkDeleteTimer(&timer);
	timer = NULL;

    result = hipMemcpy(aes_block_array, cuda_aes_block_array, block_number * sizeof(aes_block), hipMemcpyDeviceToHost);
    if (result != hipSuccess)
	{
		printf("\nDevice To Host Data Copy Failed For cuda_aes_block_array ... Exiting Now !!!\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

    for (int i = 0; i < block_number - 1; i++)
        print_file_data(aes_block_array[i].block, block_length, encFile, 1);
    print_file_data(aes_block_array[block_number - 1].block, block_length, encFile, 1);

	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

    aes_decrypt <<< BlocksPerGrid, ThreadsPerBlock >>>(cuda_aes_block_array, cuda_key, expanded_key_length, block_number);

    sdkStopTimer(&timer);
	float timeToDecrypt = sdkGetTimerValue(&timer);
	printf("\nTime To Decrypt Image on %s = %.6f seconds\n", dev_prop.name, timeToDecrypt);
	sdkDeleteTimer(&timer);
	timer = NULL;
    
    result = hipMemcpy(aes_block_array, cuda_aes_block_array, block_number * sizeof(aes_block), hipMemcpyDeviceToHost);
    if (result != hipSuccess)
	{
		printf("\nDevice To Host Data Copy Failed For cuda_aes_block_array ... Exiting Now !!!\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

    for (int i = 0; i < block_number - 1; i++)
        print_file_data(aes_block_array[i].block, block_length, decFile, 2);

    if (num_zero_pending == 0)
        print_file_data(aes_block_array[block_number - 1].block, block_length, decFile, 2);
    else
        print_file_data(aes_block_array[block_number - 1].block, block_length, decFile, 3);

	return SUCCESS;
}

void cleanup(void)
{
    if (cuda_key)
    {
        hipFree(cuda_key);
        cuda_key = NULL;
    }

    if (cuda_aes_block_array)
    {
        hipFree(cuda_aes_block_array);
        cuda_aes_block_array = NULL;
    }

	if (aes_block_array)
	{
		free(aes_block_array);
		aes_block_array = NULL;
	}

    if (decFile)
	{
		fclose(decFile);
		decFile = NULL;
	}

	if (encFile)
	{
		fclose(encFile);
		encFile = NULL;
	}

	if (key_file_stream.is_open())
		key_file_stream.close();

	if (input_file_stream.is_open())
		input_file_stream.close();
}


int main(int argc, char* argv[])
{
	// Variable Declarations
	int result;

	// Code
	print_cuda_device_properties();

	result = get_file_data(argv);
	if (result == FAILURE)
	{
		printf("Failed To Access Input Files ... Exiting Now !!!\n");
		exit(EXIT_FAILURE);
	}

	cleanup();

	return 0;
}
